/*
 * Diana-Andreea Popescu, EPFL & CERN, Switerland. All rights reserved.
 */


#include <stdio.h>
#include <assert.h>
#include <math.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "Comp.h"
// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/remove.h>
#include <omp.h>
#include <vector>
#include <list>
#include <algorithm>
#include <numeric>

using namespace std;

inline
void checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n",
            hipGetErrorString(result));
        exit(EXIT_FAILURE);
  }
}


unsigned int nextPow2(unsigned int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}


struct is_order_less
{
    __host__ __device__
    bool operator() (const int x)
    {
	    return (x < 0);
    }
};

extern "C"
int composeOnGPU(vector<double> input_coeff, vector<list<int> > terms, int inputSize, 
		 vector<int*> other_exp, vector<double*> other_coeff, vector<uint> otherSize, int order)
{
    int nr_terms = terms.size();
    int nr_functions = otherSize.size();
    hipStream_t streams[nr_terms];
    for (int i = 0; i < nr_terms; ++i) 
	    hipStreamCreate(&streams[i]); 
    //duplicate the input polynoms for each stream
    //allocate device memory for input
    ////////////////////////////////////////////
    int *e_input;
    double *c_input;
    unsigned int mem_size_exp_input = 0;
    unsigned int mem_size_coeff_input = 0;
    int all_size = accumulate(otherSize.begin(), otherSize.end(), 0);
    mem_size_exp_input = NRVARS * all_size * nr_terms * sizeof(int);
    mem_size_coeff_input = all_size * nr_terms * sizeof(double);

    checkCuda(hipMalloc((void **) &e_input, mem_size_exp_input));
    checkCuda(hipMalloc((void **) &c_input, mem_size_coeff_input));
	
    ////////////////////////////////////////////
    //determine maximum size to use for allocation 
    //allocate memory for multiplication result, aux result, stencil, indices on each stream
    unsigned int max_size = *(max_element(otherSize.begin(), otherSize.end()));
    unsigned int result_size = max_size * max_size * max_size;
    unsigned int mem_size_result_exp = result_size * NRVARS * sizeof(int) * nr_terms;
    unsigned int mem_size_result_coeff = result_size  * sizeof(double) * nr_terms;
    int *e_result, *e_aux;
    double *c_result, *c_aux;
    checkCuda(hipMalloc((void **) &e_result, mem_size_result_exp));
    checkCuda(hipMalloc((void **) &c_result, mem_size_result_coeff));
    checkCuda(hipMalloc((void **) &e_aux, mem_size_result_exp));
    checkCuda(hipMalloc((void **) &c_aux, mem_size_result_coeff));
   
    //determine next power of 2 for stencil for stream compactation
    unsigned int stencil_size =  max_size * max_size * max_size;
    unsigned int mem_size_stencil = stencil_size * sizeof(unsigned int) * nr_terms;
    int* stencil;
    checkCuda(hipMalloc((void **) &stencil, mem_size_stencil));
    //alloc memoy for keys
    unsigned long long *e_keys;
    unsigned int mem_size_keys = result_size * sizeof(unsigned long long) * nr_terms;
    checkCuda(hipMalloc((void **) &e_keys, mem_size_keys));
    ///////////////////////////////////////////////
    //copy data to GPU
    for (int i = 0; i < nr_terms; ++i){
	    //compute offset
	    int offset_exp = i * all_size * NRVARS;
	    int offset_coeff = i * all_size;
	    for (int j = 0; j < nr_functions; ++j){ 
		    //compute offset
		    if (j != 0){
			    offset_exp += otherSize[j - 1] * NRVARS; 
			    offset_coeff += otherSize[j - 1];
		    }  
		    int mem_size_exp_j = otherSize[j] * NRVARS * sizeof(int);
		    int mem_size_coeff_j = otherSize[j] * sizeof(double);
		    checkCuda(hipMemcpyAsync(e_input + offset_exp, other_exp[j], 
			    mem_size_exp_j, hipMemcpyHostToDevice, streams[i]));
		    checkCuda(hipMemcpyAsync(c_input + offset_coeff, other_coeff[j], mem_size_coeff_j, 
			    hipMemcpyHostToDevice, streams[i]));
	    }	
    }
    ////////////////////////////////////////////
    //compute number of multiplications 		
    int iterations = -nr_terms;
    for (vector<list<int> >:: const_iterator it = terms.begin(); it != terms.end(); ++it)
	    iterations += it->size();
    unsigned int index_result_exp, index_result_coeff, index_input_exp, index_input_coeff;
    vector<pair<int*, double*> > p_result;
    vector<pair<int*, double*> > p_aux;
    for (int i = 0; i < nr_terms; i ++){
	index_result_exp = i * result_size * NRVARS;
	index_result_coeff = i * result_size;
      	p_result.push_back(make_pair(e_result + index_result_exp, c_result + index_result_coeff));
    } 
     
    vector<int> single_terms;
    vector<uint> sizes_aux;
    for (int i = 0; i < nr_terms; i ++){
    	index_input_exp = i * all_size * NRVARS;
	index_input_coeff = i * all_size;
	int index = terms[i].front();
	terms[i].pop_front();
	if (terms[i].empty())
	  	single_terms.push_back(i);
	for (int k = 1; k <= index; k ++){
		 index_input_exp += otherSize[k - 1] * NRVARS;
		 index_input_coeff += otherSize[k - 1];
	}
	sizes_aux.push_back(otherSize[index]);
	p_aux.push_back(make_pair(e_input + index_input_exp, c_input + index_input_coeff));
    }
    
    vector<bool> firstmult(nr_terms, true);
    vector<int> indices;
    vector<uint> dimRes(nr_terms, 0);
    
    thrust::device_ptr<int> stencil_ptr(stencil); 
    thrust::device_ptr<double>  end_coeffs_ptr;  
    thrust::device_ptr<unsigned long long> e_keys_ptr(e_keys);

    for (int i = 0; i < single_terms.size(); i ++)
      	get_keys(e_keys + single_terms[i] * result_size, p_aux[single_terms[i]].first, sizes_aux[single_terms[i]], streams[single_terms[i]]);
    hipDeviceSynchronize();
    unsigned int dimResult, dimInput, dimAux;
    while (iterations > 0){
	    //launch kernels from each term
      //   #pragma omp parallel for private(index_input_exp, index_input_coeff, dimInput, dimAux, dimResult) shared(iterations, indices, terms, dimRes, p_aux, p_result, stencil, order, streams, input_coeff) num_threads(nr_terms)
		double start = omp_get_wtime();    
	for (int i = 0; i < nr_terms; i ++){
		    if (!terms[i].empty()) {
		      //      	#pragma omp critical
		      	indices.push_back(i);
			int index = terms[i].front();
			index_input_exp = i * all_size * NRVARS;
			index_input_coeff = i * all_size;
			for (int k = 1; k <= index; k ++){
				index_input_exp += otherSize[k - 1] * NRVARS;
				index_input_coeff += otherSize[k - 1];
			}
			
			dimInput = otherSize[index];
			dimAux = sizes_aux[i];
			dimResult = dimInput * dimAux;
			dimRes[i] = dimResult; 
			if (terms[i].size() == 1){ 
			 	 multiply_truncate_key2(p_result[i].first, e_keys + i * result_size,
			    		p_aux[i].first, e_input + index_input_exp,
			    		p_result[i].second, p_aux[i].second, c_input + index_input_coeff, 
				  	dimResult, dimAux, dimInput, order, stencil + i * stencil_size, input_coeff[i], streams[i]); 
			} 
			else {	
				multiply_truncate2(p_result[i].first, p_aux[i].first, e_input + index_input_exp,
			 	p_result[i].second, p_aux[i].second, c_input + index_input_coeff, 
				 dimResult, dimAux, dimInput, order, stencil + i * stencil_size, streams[i]); 	 			    
			}
		
		
			terms[i].pop_front();
			//      	#pragma omp critical
			iterations --;
	     	}		
	    }
	    hipDeviceSynchronize();
	    double end = omp_get_wtime();
	    cout << 1000 * (end - start) << endl;
	    double startt = omp_get_wtime();
	    //truncate
	    for (int i = 0; i < indices.size(); i ++){
	    	if (firstmult[indices[i]]){
			firstmult[indices[i]] = false;
			index_result_exp = indices[i] * result_size * NRVARS;
			index_result_coeff = indices[i] * result_size;
			p_aux[indices[i]] = make_pair(e_aux + index_result_exp, c_aux + index_result_coeff);	
		} 
		thrust::device_ptr<double> result_coeff_ptr(p_result[indices[i]].second);
		thrust::device_ptr<double> aux_coeff_ptr(p_aux[indices[i]].second);
		end_coeffs_ptr = thrust::remove_copy_if(result_coeff_ptr, result_coeff_ptr + dimRes[indices[i]], stencil_ptr + indices[i] * stencil_size, aux_coeff_ptr, is_order_less());
		int esize = end_coeffs_ptr - aux_coeff_ptr;
		//	cout << esize  << endl;
		sizes_aux[indices[i]] = esize;
  
	    	if (!terms[indices[i]].empty()){
		  	//remove if on exponents
		  	thrust::device_ptr<int> result_exp_ptr(p_result[indices[i]].first);
	       		thrust::device_ptr<int> aux_exp_ptr(p_aux[indices[i]].first);
		      	
			for (int k = 0; k < NRVARS; ++k){
			      thrust::remove_copy_if(result_exp_ptr + k * dimRes[indices[i]], 
						     result_exp_ptr + (k + 1) * dimRes[indices[i]], 
			     stencil_ptr + indices[i] * stencil_size, aux_exp_ptr + k * esize, is_order_less());
			}
		}
		else {
		  	//remove if on keys
		      thrust::remove_if(e_keys_ptr + indices[i] * result_size, e_keys_ptr + indices[i] * result_size + dimRes[indices[i]], stencil_ptr + indices[i] * stencil_size, is_order_less());	
		}	    
	  }
	    double endt = omp_get_wtime();
	    cout << "truncate " << 1000 * (endt - startt) << endl;
	  indices.clear();
    }
    double startcopy = omp_get_wtime();
    unsigned long long *keys;
    uint size_keys = accumulate(sizes_aux.begin(),sizes_aux.end(), 0);
    uint mem_size_final_keys = sizeof(unsigned long long) * size_keys;
    cout << "total " << size_keys << endl;
    checkCuda(hipMalloc((void **) &keys, mem_size_final_keys));
    thrust::device_ptr<unsigned long long> keys_ptr(keys);
    int index_keys = 0;
    thrust::device_ptr<double> c_result_ptr(c_result);
    thrust::device_ptr<double> c_aux_ptr(c_aux);
   
    for (int i = 0; i < nr_terms; i ++){
      	thrust::copy(e_keys_ptr + i * result_size, e_keys_ptr + i * result_size + sizes_aux[i], keys_ptr + index_keys);
    	thrust::copy(c_aux_ptr + i * result_size, c_aux_ptr + i * result_size + sizes_aux[i], c_result_ptr + index_keys);
	index_keys += sizes_aux[i];
    }
   
    double endcopy = omp_get_wtime();
    cout << "copy" << 1000 * (endcopy - startcopy) << endl;
    double startsort = omp_get_wtime();
    thrust::sort_by_key(keys_ptr, keys_ptr + size_keys, c_result_ptr);
    double endsort = omp_get_wtime();
    cout << "reduce" << 1000 * (endsort - startsort) << endl;
    //reduce by key 
    double startreduce = omp_get_wtime();
    thrust::pair<thrust::device_ptr<unsigned long long>, thrust::device_ptr<double> > end;
    end = thrust::reduce_by_key(keys_ptr, keys_ptr + size_keys, c_result_ptr, e_keys_ptr, c_aux_ptr); 
    size_keys = end.first - e_keys_ptr;
    double endreduce = omp_get_wtime();
    cout << "sort" << 1000 * (endreduce - startreduce) << endl;
    cout << "size " << size_keys << endl;
    get_exponents(e_result, e_keys, size_keys);		

    for (int i = 0; i < nr_terms; ++i)
	    hipStreamDestroy(streams[i]);
 	
    //free memory
    hipFree(e_input);
    hipFree(c_input);
    hipFree(e_result);
    hipFree(c_result);
    hipFree(e_aux);
    hipFree(c_aux);
    hipFree(e_keys);
    hipFree(keys);
 	
    return size_keys; 
}
